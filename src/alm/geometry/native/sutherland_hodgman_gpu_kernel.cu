#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <optional>
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>

#include "sutherland_hodgman.hpp"

using namespace std;
using namespace torch;

constexpr int num_threads = 2 * C10_WARP_SIZE;
constexpr int thread_work_size = 1024;
constexpr int block_work_size = num_threads * thread_work_size;

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

template <typename scalar_t>
__global__ void sutherland_hodgman_gpu_kernel(scalar_t *result_data,
                                              const scalar_t *poly1_data, const scalar_t *poly2_data,
                                              const int out_count, const int out_len,
                                              const int poly1_len, const int poly2_len) {
   int index = thread_work_size * (blockIdx.x * blockDim.x + threadIdx.x);
   const int end = min(index + thread_work_size, out_count);

   if (index >= end) {
      return ;
   }

   scalar_t * tmp = reinterpret_cast<scalar_t *>(malloc(2 * out_len * sizeof(scalar_t)));
   
   assert(tmp != NULL);

   const int result_stride = out_len * 2;
   const int poly1_stride = poly1_len * 2;
   const int poly2_stride = poly2_len * 2;
   int64_t npoly;

   result_data += index*result_stride;
   poly1_data += index*poly1_stride;
   poly2_data += index*poly2_stride;

   for (; index < end; ++index) {
      npoly = polygon_clip(result_data, tmp, poly1_data, poly2_data, poly1_len, poly2_len, out_len);
      
      assert(npoly <= out_len);

      result_data += result_stride;
      poly1_data += poly1_stride;
      poly2_data += poly2_stride;
   }

   free(reinterpret_cast<void *>(tmp));
}

template <typename scalar_t>
__global__ void compute_intersection_area_gpu_kernel(scalar_t *result_data,
      const scalar_t *poly1_data, const scalar_t *poly2_data,
      const int out_count, const int out_len,
      const int poly1_len, const int poly2_len) {
   int index = thread_work_size * (blockIdx.x * blockDim.x + threadIdx.x);
   const int end = min(index + thread_work_size, out_count);
   if (index >= end) {
      return ;
   }

   scalar_t * tmp = reinterpret_cast<scalar_t *>(malloc(4 * out_len * sizeof(scalar_t)));

   assert(tmp != NULL);

   const int poly1_stride = poly1_len*2;
   const int poly2_stride = poly2_len*2;
   int64_t npoly;

   result_data += index;
   poly1_data += index*poly1_stride;
   poly2_data += index*poly2_stride;

   for (; index < end; ++index) {
      npoly = polygon_clip(tmp, tmp + 2*out_len, poly1_data, poly2_data, poly1_len, poly2_len, out_len);

      assert(npoly <= out_len);

      *(result_data++) = shoelace(tmp, npoly);

      poly1_data += poly1_stride;
      poly2_data += poly2_stride;
   }

   free(reinterpret_cast<void *>(tmp));
}

Tensor sutherland_hodgman_gpu(const Tensor &poly1, const Tensor &poly2) {
   CHECK_INPUT(poly1);
   CHECK_INPUT(poly2);
   CHECK_INPUT_POLY_AND_PREPARE(poly1, poly2);

   out_shape.push_back(out_len);
   out_shape.push_back(2);
   torch::Tensor result = at::zeros(out_shape, poly1.options());

   AT_DISPATCH_FLOATING_TYPES_AND_HALF(result.scalar_type(), "sutherland_hodgman_gpu", [&] {
      const dim3 blocks((out_count + block_work_size - 1) / block_work_size);
      auto stream = at::cuda::getCurrentCUDAStream();
      sutherland_hodgman_gpu_kernel<<<blocks, num_threads, 0, stream>>>(
         result.data_ptr<scalar_t>(),
         poly1.data_ptr<scalar_t>(),
         poly2.data_ptr<scalar_t>(),
         out_count,
         out_len,
         poly1_len,
         poly2_len);
   });

   return result;
}


Tensor compute_intersection_area_gpu(const Tensor &poly1, const Tensor &poly2) {
   CHECK_INPUT(poly1);
   CHECK_INPUT(poly2);
   CHECK_INPUT_POLY_AND_PREPARE(poly1, poly2);

   torch::Tensor result = at::empty(out_shape, poly1.options());

   AT_DISPATCH_FLOATING_TYPES_AND_HALF(result.scalar_type(), "compute_intersection_area_gpu", [&] {
      const dim3 blocks((out_count + block_work_size - 1) / block_work_size);
      auto stream = at::cuda::getCurrentCUDAStream();
      compute_intersection_area_gpu_kernel<<<blocks, num_threads, 0, stream>>>(
         result.data_ptr<scalar_t>(),
         poly1.data_ptr<scalar_t>(),
         poly2.data_ptr<scalar_t>(),
         out_count,
         out_len,
         poly1_len,
         poly2_len);
   });

   return result;
}

